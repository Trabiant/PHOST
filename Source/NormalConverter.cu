#include "hip/hip_runtime.h"
#include "NormalConverter.h"
#include <hip/hip_runtime.h>
#include ""
#include <cmath>

__global__ void CamToObjectKernel(
    const uchar3* inNormal, uchar3* outNormal,
    int width, int height, Mat3x3 rotM)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;

    double nx = inNormal[idx].z / 127.5 - 1.0;
    double ny = inNormal[idx].y / 127.5 - 1.0;
    double nz = inNormal[idx].x / 127.5 - 1.0;

    double ox = rotM.m[0] * nx + rotM.m[1] * ny + rotM.m[2] * nz;
    double oy = rotM.m[3] * nx + rotM.m[4] * ny + rotM.m[5] * nz;
    double oz = rotM.m[6] * nx + rotM.m[7] * ny + rotM.m[8] * nz;

    double len = sqrt(ox * ox + oy * oy + oz * oz);
    if (len > 1e-6) { ox /= len; oy /= len; oz /= len; }
    else { ox = 0; oy = 0; oz = 1; }

    uchar3 out;
    float vx = (oz * 0.5f + 0.5f) * 255.0f;
    float vy = (-oy * 0.5f + 0.5f) * 255.0f;
    float vz = (ox * 0.5f + 0.5f) * 255.0f;
    out.x = static_cast<unsigned char>(fminf(fmaxf(vx, 0.0f), 255.0f));
    out.y = static_cast<unsigned char>(fminf(fmaxf(vy, 0.0f), 255.0f));
    out.z = static_cast<unsigned char>(fminf(fmaxf(vz, 0.0f), 255.0f));
    outNormal[idx] = out;
}

void NormalConverter::convertGPU(
    const cv::Mat& src, cv::Mat& dst) const
{
    int width = src.cols, height = src.rows;
    dst.create(height, width, src.type());

    size_t buf = width * height * sizeof(uchar3);
    uchar3* d_in = nullptr, * d_out = nullptr;
    hipMalloc(&d_in, buf);
    hipMalloc(&d_out, buf);
    hipMemcpy(d_in, src.data, buf, hipMemcpyHostToDevice);

    dim3 block(16, 16), grid((width + 15) / 16, (height + 15) / 16);
    CamToObjectKernel <<<grid, block >>> (d_in, d_out, width, height, rotPOD_);
    hipDeviceSynchronize();

    hipMemcpy(dst.data, d_out, buf, hipMemcpyDeviceToHost);
    hipFree(d_in); hipFree(d_out);
}
