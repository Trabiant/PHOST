#include "hip/hip_runtime.h"
#include "PhotometricStereo.h"
#include ""
#include <vector>
#include <algorithm>
#include <cmath>
#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>

__global__ void ComputeNormalKernel(
	const float* __restrict__ d_img_data,
	const uchar* __restrict__ d_mask,
	const float* __restrict__ d_Lpinv,
	uchar* d_out,
	int M, int h, int w)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= w || y >= h) return;
	int idx = y * w + x;
	if (d_mask[idx] > 127) return;

	float g0 = 0, g1 = 0, g2 = 0;
#pragma unroll
	for (int i = 0; i < M; ++i) {
		float v = d_img_data[size_t(i) * h * w + idx];
		g0 += d_Lpinv[0 * M + i] * v;
		g1 += d_Lpinv[1 * M + i] * v;
		g2 += d_Lpinv[2 * M + i] * v;
	}

	float nrm = sqrtf(g0 * g0 + g1 * g1 + g2 * g2);
	float nx = 0, ny = 0, nz = 1;
	if (nrm > 1e-6f) { nx = g0 / nrm; ny = g1 / nrm; nz = g2 / nrm; }

	int o = idx * 3;
	d_out[o + 0] = uchar((nz + 1) * 0.5f * 255);
	d_out[o + 1] = uchar((ny + 1) * 0.5f * 255);
	d_out[o + 2] = uchar((nx + 1) * 0.5f * 255);
}



void PhotometricStereo::ComputeNormal_GPU() {
	int M = images_.size(), h = images_[0].rows, w = images_[0].cols;
	size_t npix = size_t(h) * w;

	cv::Mat l_pinv;
	cv::invert(lightDirs_, l_pinv, cv::DECOMP_SVD);
	std::vector<float> h_Lpinv(3 * M);
	for (int i = 0; i < 3; ++i)
		for (int j = 0; j < M; ++j)
			h_Lpinv[i * M + j] = l_pinv.at<float>(i, j);

	std::vector<float> h_img_data(M * npix);
	for (int i = 0; i < M; ++i) {
		const float* ptr = images_[i].ptr<float>(0);
		std::copy(ptr, ptr + npix, h_img_data.begin() + i * npix);
	}

	float* d_img_data = nullptr, * d_Lpinv = nullptr;
	uchar* d_mask = nullptr, * d_out = nullptr;
	hipMalloc(&d_img_data, M * npix * sizeof(float));
	hipMalloc(&d_Lpinv, 3 * M * sizeof(float));
	hipMalloc(&d_mask, npix * sizeof(uchar));
	hipMalloc(&d_out, 3 * npix * sizeof(uchar));

	hipMemcpy(d_img_data, h_img_data.data(), M * npix * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_Lpinv, h_Lpinv.data(), 3 * M * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_mask, mask_.ptr<uchar>(0), npix * sizeof(uchar), hipMemcpyHostToDevice);

	dim3 block(16, 16), grid((w + 15) / 16, (h + 15) / 16);
	size_t shmem = M * sizeof(float);
    ComputeNormalKernel << <grid, block, shmem >> > (d_img_data, d_mask, d_Lpinv, d_out, M, h, w);
	hipDeviceSynchronize();

	normalMap_.create(h, w, CV_8UC3);
	hipMemcpy(normalMap_.ptr<uchar>(0), d_out, 3 * npix * sizeof(uchar), hipMemcpyDeviceToHost);

	hipFree(d_img_data);
	hipFree(d_Lpinv);
	hipFree(d_mask);
	hipFree(d_out);

}
